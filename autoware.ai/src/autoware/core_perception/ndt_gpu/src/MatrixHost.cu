#include "hip/hip_runtime.h"
#include "ndt_gpu/MatrixHost.h"
#include "ndt_gpu/debug.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "rubis_sched/sched.hpp"

namespace gpu {

MatrixHost::MatrixHost()
{
	fr_ = false;
}

MatrixHost::MatrixHost(int rows, int cols) {
	rows_ = rows;
	cols_ = cols;
	offset_ = 1;

	buffer_ = (double*)malloc(sizeof(double) * rows_ * cols_ * offset_);
	memset(buffer_, 0, sizeof(double) * rows_ * cols_ * offset_);
	fr_ = true;
}

MatrixHost::MatrixHost(int rows, int cols, int offset, double *buffer)
{
	rows_ = rows;
	cols_ = cols;
	offset_ = offset;
	buffer_ = buffer;
	fr_ = false;
}

MatrixHost::MatrixHost(const MatrixHost& other) {
	rows_ = other.rows_;
	cols_ = other.cols_;
	offset_ = other.offset_;
	fr_ = other.fr_;

	if (fr_) {
		buffer_ = (double*)malloc(sizeof(double) * rows_ * cols_ * offset_);
		memcpy(buffer_, other.buffer_, sizeof(double) * rows_ * cols_ * offset_);
	} else {
		buffer_ = other.buffer_;
	}
}

extern "C" __global__ void copyMatrixDevToDev(MatrixDevice input, MatrixDevice output) {
	int row = threadIdx.x;
	int col = threadIdx.y;
	int rows_num = input.rows();
	int cols_num = input.cols();

	if (row < rows_num && col < cols_num)
		output(row, col) = input(row, col);
}

bool MatrixHost::moveToGpu(MatrixDevice output) {
	if (rows_ != output.rows() || cols_ != output.cols())
		return false;

	if (offset_ == output.offset()) {
		rubis::sched::request_gpu();
		checkCudaErrors(hipMemcpy(output.buffer(), buffer_, sizeof(double) * rows_ * cols_ * offset_, hipMemcpyHostToDevice));
		rubis::sched::yield_gpu("8_htod");
		return true;
	}
	else {
		double *tmp;

		rubis::sched::request_gpu();
		checkCudaErrors(hipMalloc(&tmp, sizeof(double) * rows_ * cols_ * offset_));
		rubis::sched::yield_gpu("9_cudaMalloc");

		rubis::sched::request_gpu();
		checkCudaErrors(hipMemcpy(tmp, buffer_, sizeof(double) * rows_ * cols_ * offset_, hipMemcpyHostToDevice));
		rubis::sched::yield_gpu("10_htod");

		MatrixDevice tmp_output(rows_, cols_, offset_, tmp);

		dim3 block_x(rows_, cols_, 1);
		dim3 grid_x(1, 1, 1);

		rubis::sched::request_gpu();
		copyMatrixDevToDev<<<grid_x, block_x>>>(tmp_output, output);
		rubis::sched::yield_gpu("11_copyMatrixDevToDev");

		checkCudaErrors(hipDeviceSynchronize());

		rubis::sched::request_gpu();
		checkCudaErrors(hipFree(tmp));
		rubis::sched::yield_gpu("12_free");

		return true;
	}
}

bool MatrixHost::moveToHost(MatrixDevice input) {
	if (rows_ != input.rows() || cols_ != input.cols())
		return false;

	if (offset_ == input.offset()) {
		rubis::sched::request_gpu();
		checkCudaErrors(hipMemcpy(buffer_, input.buffer(), sizeof(double) * rows_ * cols_ * offset_, hipMemcpyDeviceToHost));
		rubis::sched::yield_gpu("13_dtoh");
		return true;
	}
	else {
		double *tmp;

		rubis::sched::request_gpu();
		checkCudaErrors(hipMalloc(&tmp, sizeof(double) * rows_ * cols_ * offset_));
		rubis::sched::yield_gpu("14_cudaMalloc");

		MatrixDevice tmp_output(rows_, cols_, offset_, tmp);

		dim3 block_x(rows_, cols_, 1);
		dim3 grid_x(1, 1, 1);

		rubis::sched::request_gpu();
		copyMatrixDevToDev << <grid_x, block_x >> >(input, tmp_output);
		rubis::sched::yield_gpu("15_copyMatrixDevToDev");

		checkCudaErrors(hipDeviceSynchronize());

		rubis::sched::request_gpu();
		checkCudaErrors(hipMemcpy(buffer_, tmp, sizeof(double) * rows_ * cols_ * offset_, hipMemcpyDeviceToHost));
		rubis::sched::yield_gpu("16_dtoh");

		rubis::sched::request_gpu();
		checkCudaErrors(hipFree(tmp));
		rubis::sched::yield_gpu("17_free");

		return true;
	}
}

MatrixHost &MatrixHost::operator=(const MatrixHost &other)
{
	rows_ = other.rows_;
	cols_ = other.cols_;
	offset_ = other.offset_;
	fr_ = other.fr_;

	if (fr_) {
		buffer_ = (double*)malloc(sizeof(double) * rows_ * cols_ * offset_);
		memcpy(buffer_, other.buffer_, sizeof(double) * rows_ * cols_ * offset_);
	} else {
		buffer_ = other.buffer_;
	}

	return *this;
}

void MatrixHost::debug()
{
	for (int i = 0; i < rows_; i++) {
		for (int j = 0; j < cols_; j++) {
			std::cout << buffer_[(i * cols_ + j) * offset_] << " ";
		}

		std::cout << std::endl;
	}

	std::cout << std::endl;
}

MatrixHost::~MatrixHost()
{
	if (fr_)
		free(buffer_);
}


SquareMatrixHost::SquareMatrixHost(int size) :
	 MatrixHost(size, size)
{

}

}
